#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ static void sayHello()
{
    int i = blockIdx.x*blockDim.x+threadIdx.x;

    printf("Hello from thread %d!\n", i);
}

int main()
{
    int grid_size;
    int tpb;

    puts("Please enter the grid size: ");
    scanf("%d", &grid_size);

    puts("\nPlease enter the threads per block: ");
    scanf("%d", &tpb);

    puts("\n");

    sayHello<<<grid_size,tpb>>>();

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}